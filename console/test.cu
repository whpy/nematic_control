#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <chrono>
#include <random>

#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include "cuComplexBinOp.h"
#include "cudaErr.h"

#define M_PI 3.14159265358979323846
// amount of kernels called, block_size must be n*32
#define BSZ 4

using namespace std;
typedef struct __session{
    int Nx;
    int Ny;
    float dx;
    float dy;
    float alphax;
    float alphay;
    float Lx;
    float Ly;
    float *kx;
    float *ky;
}session;

void session_init(session *s, int Nx, int Ny, float Lx, float Ly, float *kx, float *ky){
    s->Nx = Nx;
    s->Ny = Ny;
    s->Lx = Lx;
    s->Ly = Ly;
    s->dx = Lx/Nx;
    s->dy = Ly/Ny;
    s->alphax = 2*M_PI/Lx;
    s->alphay = 2*M_PI/Ly;
    s->kx = kx;
    s->ky = ky;
}
int Nx = 8;
int Ny = 8;
float Lx = 2*M_PI;
float Ly = 2*M_PI;
float dx = 2*M_PI/Nx;
float dy = 2*M_PI/Ny;
dim3 dimGrid  (int((Nx-0.5)/BSZ) + 1, int((Ny-0.5)/BSZ) + 1);
dim3 dimBlock (BSZ, BSZ); 

class session{
    public:
        int Nx;
        int Ny;

        float Lx;
        float dx;
        float alphax;

        float Ly;
        float dy;
        float alphay;

        session(int Nx, int Ny, float Lx, float Ly):Nx(Nx), Ny(Ny), Lx(Lx),Ly(Ly){
            dx = Lx/Nx;
            alphax = Lx/
            dy = Ly/Ny;

        }

};
//cufft_error_func( hipfftPlan2d( &transf, Ny, Nx, HIPFFT_R2C ) );
//cufft_error_func( hipfftPlan2d( &inv_transf, Ny, Nx, HIPFFT_C2R ) );

__global__ void D_init(float *t, int Nx, int Ny, float dx, float dy){
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nx + i;
    if(i < Nx && j < Ny){
        t[index] = sin((float)j*dy);
    }
}

__global__ void exact(float *t, int Nx, int Ny, float dx, float dy){
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nx + i;
    if(i < Nx && j < Ny){
        t[index] = cos((float)j*dy);
    }
}
void init(float *t){
    D_init<<<dimGrid, dimBlock>>>(t,Nx,Ny,dx,dy);
}


void print_float(float* t, int Nx, int Ny) {
    for (int j = 0; j < Ny; j++) {
        for (int i = 0; i < Nx; i++) {
            cout <<t[j*Nx+i] << ",";
        }
        cout << endl;
    }
}

void transforwards(float *f, hipComplex *ft)
{
    hipfftHandle plan;
    hipfftPlan2d(&plan, Nx, Ny,HIPFFT_C2C);
    hipfftExecC2C(plan, f, ft, HIPFFT_FORWARD);
} 

void transbackwards(hipComplex *ft, float *f){
    hipfftHandle plan;
    hipfftPlan2d(&plan, Nx, Ny,HIPFFT_C2C);
    hipfftExecC2C(plan, ft, f, HIPFFT_BACKWARD);
    ComplextoReal(f)
}
__global__ void Dx(hipComplex *ft, hipComplex *dft, float* kx, int Nx, int Ny){
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nx + i;

    dft[index] = ft[index]*im()*kx[i];
}

__global__ void Dy(hipComplex *ft, hipComplex *dft, float* ky, int Nx, int Ny){
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nx + i;

    dft[index] = ft[index]*im()*ky[j];
}

__global__ void RealtoComplex(float *f, hipComplex *fc, int Nx, int Ny){
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nx + i;
    fc[index].x = f[index];
    fc[index].y = 0;
}

__global__ void ComplextoReal(hipComplex *fc, float *f, int Nx, int Ny){
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nx + i;
    f[index] = fc[index].x/(Nx*Ny);
}

int main(){
    float *test;
    float *exe;
    hipComplex* ctest;
    hipComplex *htest;
    hipComplex *tmp;
    float *kx;
    float *ky;
    cuda_error_func(hipMallocManaged( &test, sizeof(float)*(Nx*Ny) ) );
    cuda_error_func(hipMallocManaged( &exe, sizeof(float)*(Nx*Ny) ) );
    cuda_error_func(hipMallocManaged( &ctest, sizeof(hipComplex)*(Nx*Ny)));
    cuda_error_func(hipMallocManaged( &htest, sizeof(hipComplex)*(Nx*Ny)));
    cuda_error_func(hipMallocManaged( &tmp, sizeof(hipComplex)*(Nx*Ny)));
    cuda_error_func(hipMallocManaged( &kx, sizeof(float)*(Nx)));
    cuda_error_func(hipMallocManaged( &ky, sizeof(float)*Ny));
    float alpha = 2*M_PI;
<<<<<<< HEAD
    for (int i = 0; i < Nx/2; i++) {
        kx[i] = i*2*M_PI/alpha;
    }
    for (int i=Nx/2+1; i<Nx; i++){
        kx[i] = (i-Nx)*2*M_PI/alpha;
    }

    for (int j=0; j<Ny/2; j++){
        ky[j] = j*2*M_PI/alpha;
    }
    for (int j=Ny/2+1; j<Ny; j++){
        ky[j] = (j-Ny)*2*M_PI/alpha;
    }
    session s;
    session_init(&s,Nx,Ny,Lx,Ly,kx,ky);
    cout << "Lx: " << s.Lx << endl <<
    "Ly: " << s.Ly << endl <<
    "Nx: " << s.Nx << endl <<
    "Ny: " << s.Ny << endl <<
    "dx: " << s.dx << endl <<
    "dy: " << s.dy << endl <<
    "alphax: " << s.alphax << endl <<
    "alphay: " << s.alphay << endl;
    hipComplex* ctest;
    
    hipfftHandle plan;
    hipfftPlan2d(&plan, Nx, Ny, HIPFFT_C2C);
    cuda_error_func(hipMallocManaged( &test, sizeof(float)*Nx*Ny));
    cuda_error_func(hipMallocManaged( &ctest, sizeof(hipComplex)*(Nx+1)/2*Ny));
=======
    for (int i=0; i<=Nx/2; i++)          
	{
	   kx[i] = i*2*M_PI/alpha;
    } 
	for (int i=Nx/2+1; i<Nx; i++)          
	{
        kx[i] = (i - Nx) * 2*M_PI/alpha;
	}

    for (int i=0; i<=Ny/2; i++)          
	{
	   ky[i] = i*2*M_PI/alpha;
    } 
	for (int i=Ny/2+1; i<Ny; i++)          
	{
        ky[i] = (i - Ny) * 2*M_PI/alpha;
	}

>>>>>>> d327c79163cf810f7a7f6bf92388042ffd7b3676
    init(test);
    exact<<<dimGrid,dimBlock>>>(exe, Nx, Ny, dx, dy);
    cuda_error_func( hipDeviceSynchronize() );
    cout << "exact" << endl;
    print_float(exe, Nx, Ny);
    cuda_error_func( hipDeviceSynchronize() );
    cout << "input: " << endl;
    print_float(test, Nx, Ny);
    RealtoComplex<<<dimGrid,dimBlock>>>(test, ctest, Nx, Ny);
    cuda_error_func( hipDeviceSynchronize() );
    cout << "here" << endl;

    hipfftHandle plan;
    hipfftPlan2d(&plan, Nx, Ny,HIPFFT_C2C);
    hipfftExecC2C(plan, ctest, htest, HIPFFT_FORWARD);
    hipfftExecC2C(plan, htest, ctest, HIPFFT_BACKWARD);
    ComplextoReal<<<dimGrid,dimBlock>>>(ctest, test, Nx, Ny);
    cuda_error_func( hipDeviceSynchronize() );
    cout << endl << "after fft and ifft" << endl;
    print_float(test, Nx, Ny);

    Dx<<<dimGrid,dimBlock>>>(htest, tmp, kx, Nx, Ny);
    cuda_error_func( hipDeviceSynchronize() );
    hipfftExecC2C(plan,tmp, ctest, HIPFFT_BACKWARD);
    ComplextoReal<<<dimGrid,dimBlock>>>(ctest, test, Nx, Ny);
    cuda_error_func( hipDeviceSynchronize() );
    cout << endl << "after derive x" << endl;
    print_float(test, Nx, Ny);

    Dy<<<dimGrid,dimBlock>>>(htest, tmp, kx, Nx, Ny);
    cuda_error_func( hipDeviceSynchronize() );
    hipfftExecC2C(plan,tmp, ctest, HIPFFT_BACKWARD);
    ComplextoReal<<<dimGrid,dimBlock>>>(ctest, test, Nx, Ny);
    cuda_error_func( hipDeviceSynchronize() );
    cout << endl << "after derive y" << endl;
    print_float(test, Nx, Ny);

    return 0;
}