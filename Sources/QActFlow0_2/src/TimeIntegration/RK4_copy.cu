#include "hip/hip_runtime.h"
#include <TimeIntegration/RK4.cuh>



//RK4 integrating steps
// du/dt = \alpha*u + F(t,u)
// IFh = exp(\alpha*dt/2). IF = exp(\alpha*dt)
// u_{n+1} = u_{n}*IF + 1/6*(a*IF + 2b*IFh + 2c*IFh + d)
// a_n = dt*F(t_n,u_n)
// b_n = dt*F(t_n+dt/2, (u_n+a_n/2)*IFh)
// c_n = dt*F(t_n+dt/2, u_n*IFh + b_n/2)
// d_n = dt*F(t_n+dt, u_n*IF + c_n*IFh)
// 
// Here, u_old represents u_n, u_curr represents the input for computing
// the intermediate variables (a,b,c,d), u_new represents the value at next
// step. 

// preparation before RK4 integration
// compute the value at next step adding the first term u_{n}*IF of the summation 
// prepare input ucurr(u_n) for computation of a_n
void integrate_func0(Field *u_old, Field *u_curr, Field *u_new, float dt, 
float *IF, float *IFh){
    Mesh* mesh = u_old->mesh;
    int Nxh = mesh->Nxh;
    int Ny = mesh->Ny;
    int BSZ = mesh->BSZ;
    dim3 dimGrid = mesh->dimGridsp;
    dim3 dimBlock = mesh->dimBlocksp; 
    // u_{n+1} = u_{n}*exp(alpha * dt)
    SpecMul<<<dimGrid, dimBlock>>>(u_old->spec, IF, 1., u_new->spec, Nxh, Ny, BSZ);
    // u_curr = u_{n}
    SpecSet<<<dimGrid, dimBlock>>>(u_curr->spec, u_old->spec, Nxh, Ny, BSZ);
}
// __global_
// void integrate_func0(hipComplex* spec_old, hipComplex* spec_curr, hipComplex* spec_new,
//                     float* IF, float* IFh, int Nxh, int Ny, int BSZ, float dt){
//     int i = blockIdx.x * BSZ + threadIdx.x;
//     int j = blockIdx.y * BSZ + threadIdx.y;
//     int index = j*Nxh + i;
//     if(i < Nxh && j < Ny){
//         // u_{n+1} = u_{n}*exp(alpha * dt)
//         spec_new[index] = spec_old[index]*IF[index];
//         // u_{n}
//         spec_curr[index] = spec_old[index];
//     }
// }
void integrate_func1(Field *u_old, Field *u_curr, Field *u_new, Field* u_nonl, float dt, 
float *IF, float *IFh){
    Mesh* mesh = u_old->mesh;
    int Nxh = mesh->Nxh;
    int Ny = mesh->Ny;
    int BSZ = mesh->BSZ;
    dim3 dimGrid = mesh->dimGridsp;
    dim3 dimBlock = mesh->dimBlocksp; 
    integrate_func1D<<<dimGrid, dimBlock>>>(u_old->spec, u_curr->spec, u_new->spec, u_nonl->spec, 
    IF, IFh, Nxh, Ny, BSZ, dt);
}
__global__ 
void integrate_func1D(hipComplex* spec_old, hipComplex* spec_curr, hipComplex* spec_new, hipComplex* spec_nonl,
            float* IF, float* IFh, int Nxh, int Ny, int BSZ, float dt){
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nxh + i;
    
    hipComplex an = spec_nonl[index]*dt;
    
    // u_{n+1} = u_{n}*exp(alpha * dt) + 1/6*exp(alpha*dt)*(a_n)
    spec_new[index] = spec_new[index] + float(1/6*IF[index]) * an;
    printf("(%f,%f),%f \n", an.x, an.y, float(1/6)*IF[index]);
    // u_curr = (u_{n}+a_{n}/2)*exp(alpha*dt/2)
    spec_curr[index] = (spec_old[index]+an/2) * IFh[index];
}


// __global__ 
// void integrate_func1(hipComplex* spec_old, hipComplex* spec_curr, hipComplex* spec_new, hipComplex* spec_nonl,
//                     float* IF, float* IFh, int Nxh, int Ny, int BSZ, float dt){
//     // spec_nonl = a_n/dt here
//     // spec_curr represents the value to be input into Nonlinear function for b_n/dt next 
//     int i = blockIdx.x * BSZ + threadIdx.x;
//     int j = blockIdx.y * BSZ + threadIdx.y;
//     int index = j*Nxh + i;
//     if(i < Nxh && j < Ny){
//         hipComplex an = spec_nonl[index]*dt;
//         // u_{n+1} = u_{n}*exp(alpha * dt) + 1/6*exp(alpha*dt)*(a_n)
//         spec_new[index] = spec_new[index] + 1/6*IF[index] * an;
//         // (u_{n}+a_{n}/2)*exp(alpha*dt/2)
//         spec_curr[index] = (spec_old[index]+an/2) * IFh[index];
//     }
// }
__global__ void integrate_func2(hipComplex* spec_old, hipComplex* spec_curr, hipComplex* spec_new, 
                        hipComplex* spec_nonl,float* IF, float* IFh, int Nxh, int Ny, int BSZ, float dt){
    // spec_nonl = b_n/dt here
    // spec_curr represents the value to be input into Nonlinear function for c_n/dt next 
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nxh + i;
    if(i < Nxh && j < Ny){
        hipComplex bn = spec_nonl[index]*dt;
        // u_{n+1} = u_{n}*exp(alpha * dt) + 1/6*exp(alpha*dt)*(a_n) + 1/6*exp(alpha*dt/2)*(b_n)
        spec_new[index] = spec_new[index] + 1/3*IFh[index] * bn;
        // (u_{n}*exp(alpha*dt/2) + b_{n}/2)
        spec_curr[index] = (spec_old[index]*IFh[index] + bn/2) ;
    }
}
__global__ void integrate_func3(hipComplex* spec_old, hipComplex* spec_curr, hipComplex* spec_new, 
                        hipComplex* spec_nonl,float* IF, float* IFh, int Nxh, int Ny, int BSZ, float dt){
    // spec_nonl = c_n/dt here
    // spec_curr represents the value to be input into Nonlinear function for d_n/dt next 
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nxh + i;
    if(i < Nxh && j < Ny){
        hipComplex cn = spec_nonl[index]*dt;
        // u_{n+1} = u_{n}*exp(alpha * dt) + 1/6*exp(alpha*dt)*(a_n) + 1/3*exp(alpha*dt/2)*(b_n) 
        //         + 1/3*exp(alpha*dt/2)*(c_n)
        spec_new[index] = spec_new[index] + 1/3*IFh[index] * cn;
        // u_{n}*exp(alpha*dt) + c_{n} * exp(alpha*dt/2)
        spec_curr[index] = (spec_old[index]*IF[index] + cn*IFh[index]) ;
    }
}
__global__ void integrate_func4(hipComplex* spec_old, hipComplex* spec_curr, hipComplex* spec_new, 
                        hipComplex* spec_nonl,float* IF, float* IFh, int Nxh, int Ny, int BSZ, float dt){
    // spec_nonl = d_n/dt here
    // spec_curr represents the value to be input into Nonlinear function for d_n/dt next 
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nxh + i;
    if(i < Nxh && j < Ny){
        hipComplex dn = spec_nonl[index]*dt;
        // u_{n+1} = u_{n}*exp(alpha * dt) + 1/6*exp(alpha*dt)*(a_n) + 1/3*exp(alpha*dt/2)*(b_n) 
        //         + 1/3*exp(alpha*dt/2)*(c_n) + 1/6*d_n
        spec_new[index] = spec_new[index] + 1/6*dn;
    }

}