#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <fstream>
#include <string>
#include <cmath>
#include <chrono>
#include <random>
#include <stdio.h>
#include "cuComplexBinOp.h"
#include "cudaErr.h"

#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include "cuComplexBinOp.h"
#define BSZ 4

using namespace std;

__global__ 
void hello1D(){
    printf("hello1 \n");
}

__global__ 
void hello2D(){
    printf("hello2 \n");
}

void hello1(){
    hello1D<<<1,16>>>();
}

void hello2(){
    hello2D<<<1,32>>>();
}

__global__ void reality_func(hipComplex *spec, int Nxh, int Ny){
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nxh + i;
    hipComplex mean_value{ 0.f, 0.f };
    if(j<Ny && i == 0){
        mean_value = 0.5f * ( spec[index] + hipConjf(spec[Nxh*Ny-index]) );
        spec[index] = mean_value;
		spec[Nxh*Ny-index] = hipConjf(mean_value);
    }
}

void print_func(hipComplex *spec, int Nxh, int Ny){
    for (int j = 0; j < Ny; j++){
        for (int i = 0; i < Nxh; i++){
            int index = j*Nxh + i;
            printf("( %f, %f), ", spec[index].x, spec[index].y);
        }
        printf("\n");
    }
}
int main(){
    hipComplex *spec;
    
    int Ny = 16;
    int Nx = 16;
    int Nxh = Nx/2+1;
    dim3 dimGrid  (int((Nx-0.5)/BSZ) + 1, int((Ny-0.5)/BSZ) + 1);
    dim3 dimBlock (BSZ, BSZ); 

    hipMallocManaged(&spec, sizeof(hipComplex)*Ny*Nxh);
    for (int i = 0; i < Nxh; i++){
        for (int j = 0; j < Ny; j++){
            int index = j*Nxh + i;
            if (i==0){
                spec[index] = make_hipComplex((float)(j)/2, (float)j);
            }
            else{
                spec[index] = make_hipComplex(0.f,0.f);
            }
        }
    }
    print_func(spec, Nxh, Ny);
    cout << "after symmetry." <<endl;
    reality_func<<<dimGrid, dimBlock>>>(spec, Nxh, Ny);
    hipDeviceSynchronize();
    print_func(spec, Nxh, Ny);
    
    return 0;
}