#include "hip/hip_runtime.h"
#include <Basic/QActFlow.h>
#include <Basic/FldOp.cuh>
#include <Field/Field.h>
#include <Stream/Streamfunc.cuh>
#include <iostream>

using namespace std;

// \phi = cos(x)*sin(y)
// w = Laplacian(\phi) = -2*cos(x)*sin(y)
// u = -1*Dy(\phi) = -1*cos(x)*cos(y)
// v = Dx(\phi) = -1*sin(x)*sin(y)
__global__
void rinitD(float* r1, float* r2, int Nx, int Ny, int BSZ, float dx, float dy){
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = i + j*Nx;
    float x = i*dx;
    float y = j*dy;
    if (i<Nx && j<Ny){
        r1[index] = cos(x+y);
        r2[index] = sin(x+y);
    }
}

// S = 2*sqrt(r1^2+r2^2) = 2
__global__
void SexactD(float* S, int Nx, int Ny, int BSZ, float dx, float dy){
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = i + j*Nx;
    float x = i*dx;
    float y = j*dy;
    if (i<Nx && j<Ny){
        S[index] = 2.f;
    }
}
// __global__
// void wexactD(float* phys, int Nx, int Ny, int BSZ, float dx, float dy){
//     int i = blockIdx.x * BSZ + threadIdx.x;
//     int j = blockIdx.y * BSZ + threadIdx.y;
//     int index = i + j*Nx;
//     float x = i*dx;
//     float y = j*dy;
//     if (i<Nx && j<Ny){
//         phys[index] = -2*cos(x)*sin(y);
//     }
// }

// __global__
// void uexactD(float* phys, int Nx, int Ny, int BSZ, float dx, float dy){
//     int i = blockIdx.x * BSZ + threadIdx.x;
//     int j = blockIdx.y * BSZ + threadIdx.y;
//     int index = i + j*Nx;
//     float x = i*dx;
//     float y = j*dy;
//     if (i<Nx && j<Ny){
//         phys[index] = -1*cos(x)*cos(y);
//     }
// }
// __global__
// void vexactD(float* phys, int Nx, int Ny, int BSZ, float dx, float dy){
//     int i = blockIdx.x * BSZ + threadIdx.x;
//     int j = blockIdx.y * BSZ + threadIdx.y;
//     int index = i + j*Nx;
//     float x = i*dx;
//     float y = j*dy;
//     if (i<Nx && j<Ny){
//         phys[index] = -1*sin(x)*sin(y);
//     }
// }
inline void init(Field* r1, Field* r2, Field* Sa){
    Mesh* mesh = r1->mesh;
    int Nx = mesh->Nx; int Ny = mesh->Ny; int BSZ = mesh->BSZ; 
    float dx = mesh->dx; float dy = mesh->dy;
    dim3 dimGrid = mesh->dimGridp; dim3 dimBlock = mesh->dimBlockp;
    rinitD<<<dimGrid, dimBlock>>>(r1->phys, r2->phys, Nx, Ny, BSZ, dx, dy);
    SexactD<<<dimGrid, dimBlock>>>(Sa->phys, Nx, Ny, BSZ, dx, dy);
    // uexactD<<<dimGrid, dimBlock>>>(ua->phys, Nx, Ny, BSZ, dx, dy);
    // vexactD<<<dimGrid, dimBlock>>>(va->phys, Nx, Ny, BSZ, dx, dy);
    // update the spectral
    // FwdTrans(mesh, r1->phys, ->spec);
}

void field_visual(Field *f, string name){
    Mesh* mesh = f->mesh;
    ofstream fval;
    string fname = name;
    fval.open(fname);
    for (int j=0; j<mesh->Ny; j++){
        for (int i=0; i<mesh->Nx; i++){
            fval << f->phys[j*mesh->Nx+i] << ",";
        }
        fval << endl;
    }
    fval.close();
}

void print_spec(Field* f){
    Mesh* mesh = f->mesh;
    int Nxh = mesh->Nxh, Ny = mesh->Ny;
    for(int j = 0; j < Ny; j++){
        for (int i = 0; i < Nxh; i++){
            int index = i + j*Nxh;
            cout << "("<< f->spec[index].x << "," << f->spec[index].y << ")" << " ";
        }
        cout << endl;
    }
}

void print_phys(Field* f){
    Mesh* mesh = f->mesh;
    int Nx = mesh->Nx, Ny = mesh->Ny;
    for(int j = 0; j < Ny; j++){
        for (int i = 0; i < Nx; i++){
            int index = i + j*Nx;
            cout << "("<< f->phys[index]<< ")" << " ";
        }
        cout << endl;
    }
}

void coord(Mesh &mesh){
    ofstream xcoord("x.csv");
    ofstream ycoord("y.csv");
    for (int j=0; j<mesh.Ny; j++){
        for ( int i=0; i< mesh.Nx; i++){
            float x = mesh.dx*i;
            float y = mesh.dy*j;
            xcoord << x << ",";
            ycoord << y << ",";
        }
        xcoord << endl;
        ycoord << endl;
    }
    xcoord.close();
    ycoord.close();
}

// we test the necessary stream function method in this file 

int main(){
    int BSZ = 16;
    int Ns = 1000;
    int Nx = 512; // same as colin
    int Ny = 512;
    int Nxh = Nx/2+1;
    float Lx = 2*M_PI;
    float Ly = 2*M_PI;
    float dx = 2*M_PI/Nx;
    float dy = 2*M_PI/Ny;
    float dt = 0.05; // same as colin
    float a = 1.0;

/////////// 2 ////////////
// we previously have verified the validity of laplacian and vel_func.
// in this file we test the func about the Q tensor (components, r1, r2) and 
// the intermediate components p (p11, p12, p21)
    Mesh *mesh = new Mesh(BSZ, Nx, Ny, Lx, Ly);
    Field *r1 = new Field(mesh); Field *r2 = new Field(mesh);
    Field *S = new Field(mesh); Field *Sa = new Field(mesh);

    // aux is the abbreviation of auxiliary, where only act as intermediate values
    // to assist computation. So we should guarantee that it doesnt undertake any 
    // long term memory work.
    Field *aux = new Field(mesh); Field *aux1 = new Field(mesh);
    // Field* phi = new Field(mesh);
    // Field* w = new Field(mesh); Field* wa = new Field(mesh);
    // Field* u = new Field(mesh); Field* ua = new Field(mesh);
    // Field* v = new Field(mesh); Field* va = new Field(mesh);

    coord(*mesh);
    init(r1, r2, Sa);
    cuda_error_func( hipDeviceSynchronize() );
    field_visual(r1, "r1.csv");
    field_visual(r2, "r2.csv");
    field_visual(Sa, "Sa.csv");
    
    // evaluate the S
    S_func(r1, r2, S);
    // S_funcD<<<mesh->dimGridp, mesh->dimBlockp>>>(r1->phys, r2->phys, S->phys, Nx, Ny, BSZ);
    cuda_error_func( hipDeviceSynchronize() );
    field_visual(S, "S.csv");
//////////////////// Sfunc tested //////////////////

//////////////////// crossfunc test ///////////////









    return 0;
}