#include "hip/hip_runtime.h"
#include <Basic/QActFlow.h>
#include <Basic/FldOp.cuh>
#include <Field/Field.h>
#include <Basic/cuComplexBinOp.h>
#include <TimeIntegration/RK4.cuh>
#include <stdlib.h>
#include <iostream>

using namespace std;
__global__ void init_func(float* fp, float dx, float dy, int Nx, int Ny, int BSZ){
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nx + i;
    if(i<Nx && j<Ny){
        float x = i*dx;
        float y = j*dy;
        fp[index] = -sin(x+y);
    }
}

void field_visual(Field *f, string name){
    Mesh* mesh = f->mesh;
    ofstream fval;
    string fname = name;
    fval.open(fname);
    for (int j=0; j<mesh->Ny; j++){
        for (int i=0; i<mesh->Nx; i++){
            fval << f->phys[j*mesh->Nx+i] << ",";
        }
        fval << endl;
    }
    fval.close();
}

// du/dt = -u = L(u) + NL(u),
// L(u) = -1*u, NL(u) = 0
__global__
void ulin_func(float* IFuh, float* IFu, float* k_squared, 
float dt, int Nxh, int Ny, int BSZ)
{
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nxh + i;
    float alpha = -1.0f;
    if(i<Nxh && j<Ny){
        IFuh[index] = exp( alpha *dt/2);
        IFu[index] = exp( alpha *dt);
    }
}

void unonl_func(Field* unonl, Field* ucurr,float t){
    Mesh* mesh = unonl->mesh;
    dim3 dimGrid = mesh->dimGridp;
    dim3 dimBlock = mesh->dimBlockp;
    // unonl = ucurr*ucurr
    FldSet<<<dimGrid, dimBlock>>>(unonl->phys, 0.f, mesh->Nx, mesh->Ny, mesh->BSZ);
    FwdTrans(unonl->mesh, unonl->phys, unonl->spec);
}

void print_spec(Field* f){
    Mesh* mesh = f->mesh;
    int Nxh = mesh->Nxh, Ny = mesh->Ny;
    for(int j = 0; j < Ny; j++){
        for (int i = 0; i < Nxh; i++){
            int index = i + j*Nxh;
            cout << "("<< f->spec[index].x << "," << f->spec[index].y << ")" << " ";
        }
        cout << endl;
    }
}

void print_phys(Field* f){
    Mesh* mesh = f->mesh;
    int Nx = mesh->Nx, Ny = mesh->Ny;
    for(int j = 0; j < Ny; j++){
        for (int i = 0; i < Nx; i++){
            int index = i + j*Nx;
            cout << "("<< f->phys[index]<< ")" << " ";
        }
        cout << endl;
    }
}

// we test the performance of the RK4 on linear ODE that du/dt = -u where
// the exact solution should be u = c0*exp(-t), c0 depends on initial conditon.
int main(){
    int BSZ = 16;
    int Ns = 1000;
    int Nx = 8; // same as colin
    int Ny = 8;
    int Nxh = Nx/2+1;
    float Lx = 2*M_PI;
    float Ly = 2*M_PI;
    float dx = 2*M_PI/Nx;
    float dy = 2*M_PI/Ny;
    float dt = 0.05; // same as colin
    float a = 1.0;

    // Fldset test
    Mesh *mesh = new Mesh(BSZ, Nx, Ny, Lx, Ly);
    Field *u = new Field(mesh);
    Field *unonl = new Field(mesh);
    Field *ucurr = new Field(mesh);
    Field *unew = new Field(mesh);
    float *IFu, *IFuh;
    hipMallocManaged(&IFu, sizeof(float)*Nxh*Ny);
    hipMallocManaged(&IFuh, sizeof(float)*Nxh*Ny);
    int m = 0;
    // initialize the field
    // set up the Integrating factor
    // we may take place here by IF class
    ulin_func<<<mesh->dimGridsp,mesh->dimBlocksp>>>(IFuh, IFu, mesh->k_squared, dt, mesh->Nxh, mesh->Ny, mesh->BSZ);
    // initialize the physical space of u(u_o.x << "," << f->phys[index].y ld)
    init_func<<<mesh->dimGridp,mesh->dimBlockp>>>(u->phys, 
    mesh->dx, mesh->dy, mesh->Nx, mesh->Ny, mesh->BSZ);
    // initialize the spectral space of u 
    FwdTrans(mesh, u->phys, u->spec);
    cuda_error_func( hipDeviceSynchronize() );
    print_phys(u);

    for(;m<Ns;m++){
        integrate_func0(u, ucurr, unew, IFu, IFuh, dt);
        BwdTrans(mesh, ucurr->spec, ucurr->phys);
        unonl_func(unonl, ucurr, m*dt);

        integrate_func1(u, ucurr, unew, unonl, IFu, IFuh, dt);
        BwdTrans(mesh, ucurr->spec, ucurr->phys);
        unonl_func(unonl, ucurr, m*dt);

        integrate_func2(u, ucurr, unew, unonl, IFu, IFuh, dt);
        BwdTrans(mesh, ucurr->spec, ucurr->phys);
        unonl_func(unonl, ucurr, m*dt);

        integrate_func3(u, ucurr, unew, unonl, IFu, IFuh, dt);
        BwdTrans(mesh, ucurr->spec, ucurr->phys);
        unonl_func(unonl, ucurr, m*dt);

        integrate_func4(u, ucurr, unew, unonl, IFu, IFuh, dt);
        BwdTrans(mesh, ucurr->spec, ucurr->phys);
        unonl_func(unonl, ucurr, m*dt);

        SpecSet<<<mesh->dimGridsp, mesh->dimBlocksp>>>(u->spec, unew->spec, mesh->Nxh, mesh->Ny, mesh->BSZ);
        if (m%20 == 0){
            BwdTrans(mesh, u->spec, u->phys);
            cout<<"t: " << m*dt << "  " << u->phys[5] << endl;
        }
    }
    
    return 0;
}