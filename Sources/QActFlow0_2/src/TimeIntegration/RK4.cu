#include "hip/hip_runtime.h"
#include <TimeIntegration/RK4.cuh>



//RK4 integrating steps
// du/dt = \alpha*u + F(t,u)
// IFh = exp(\alpha*dt/2). IF = exp(\alpha*dt)
// u_{n+1} = u_{n}*IF + 1/6*(a*IF + 2b*IFh + 2c*IFh + d)
// a = dt*F(t_n,u_n)
// b = dt*F(t_n+dt/2, (u_n+a/2)*IFh)
// c = dt*F(t_n+dt/2, u_n*IFh + b/2)
// d = dt*F(t_n+dt, u_n*IF + c*IFh)
__global__
void integrate_func0(hipComplex* spec_old, hipComplex* spec_curr, hipComplex* spec_new,
                    float* IF, float* IFh, int Nxh, int Ny, int BSZ, float dt){
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nxh + i;
    if(i < Nxh && j < Ny){
        // u_{n+1} = u_{n}*exp(alpha * dt)
        spec_new[index] = spec_old[index]*IF[index];
        // u_{n}
        spec_curr[index] = spec_old[index];
    }
}
__global__ 
void integrate_func1(hipComplex* spec_old, hipComplex* spec_curr, hipComplex* spec_new, hipComplex* spec_nonl,
                    float* IF, float* IFh, int Nxh, int Ny, int BSZ, float dt){
    // spec_nonl = a_n/dt here
    // spec_curr represents the value to be input into Nonlinear function for b_n/dt next 
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nxh + i;
    if(i < Nxh && j < Ny){
        hipComplex an = spec_nonl[index]*dt;
        // u_{n+1} = u_{n}*exp(alpha * dt) + 1/6*exp(alpha*dt)*(a_n)
        spec_new[index] = spec_new[index] + 1.f/6.f*IF[index] * an;
        // (u_{n}+a_{n}/2)*exp(alpha*dt/2)
        spec_curr[index] = (spec_old[index]+an/2.f) * IFh[index];
    }
}
__global__ void integrate_func2(hipComplex* spec_old, hipComplex* spec_curr, hipComplex* spec_new, 
                        hipComplex* spec_nonl,float* IF, float* IFh, int Nxh, int Ny, int BSZ, float dt){
    // spec_nonl = b_n/dt here
    // spec_curr represents the value to be input into Nonlinear function for c_n/dt next 
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nxh + i;
    if(i < Nxh && j < Ny){
        hipComplex bn = spec_nonl[index]*dt;
        // u_{n+1} = u_{n}*exp(alpha * dt) + 1/6*exp(alpha*dt)*(a_n) + 1/6*exp(alpha*dt/2)*(b_n)
        spec_new[index] = spec_new[index] + 1.f/3.f*IFh[index] * bn;
        // (u_{n}*exp(alpha*dt/2) + b_{n}/2)
        spec_curr[index] = (spec_old[index]*IFh[index] + bn/2.f) ;
    }
}
__global__ void integrate_func3(hipComplex* spec_old, hipComplex* spec_curr, hipComplex* spec_new, 
                        hipComplex* spec_nonl,float* IF, float* IFh, int Nxh, int Ny, int BSZ, float dt){
    // spec_nonl = c_n/dt here
    // spec_curr represents the value to be input into Nonlinear function for d_n/dt next 
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nxh + i;
    if(i < Nxh && j < Ny){
        hipComplex cn = spec_nonl[index]*dt;
        // u_{n+1} = u_{n}*exp(alpha * dt) + 1/6*exp(alpha*dt)*(a_n) + 1/3*exp(alpha*dt/2)*(b_n) 
        //         + 1/3*exp(alpha*dt/2)*(c_n)
        spec_new[index] = spec_new[index] + 1.f/3.f*IFh[index] * cn;
        // u_{n}*exp(alpha*dt) + c_{n} * exp(alpha*dt/2)
        spec_curr[index] = (spec_old[index]*IF[index] + cn*IFh[index]) ;
    }
}
__global__ void integrate_func4(hipComplex* spec_old, hipComplex* spec_curr, hipComplex* spec_new, 
                        hipComplex* spec_nonl,float* IF, float* IFh, int Nxh, int Ny, int BSZ, float dt){
    // spec_nonl = d_n/dt here
    // spec_curr represents the value to be input into Nonlinear function for d_n/dt next 
    int i = blockIdx.x * BSZ + threadIdx.x;
    int j = blockIdx.y * BSZ + threadIdx.y;
    int index = j*Nxh + i;
    if(i < Nxh && j < Ny){
        hipComplex dn = spec_nonl[index]*dt;
        // u_{n+1} = u_{n}*exp(alpha * dt) + 1/6*exp(alpha*dt)*(a_n) + 1/3*exp(alpha*dt/2)*(b_n) 
        //         + 1/3*exp(alpha*dt/2)*(c_n) + 1/6*d_n
        spec_new[index] = spec_new[index] + 1.f/6.f*dn;
    }

}